#include <cstdio>
#include <hipblas.h>

#include "matmul.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

#define CHECK_CUBLAS(call)  \
  do {  \
    hipblasStatus_t status_ = call;  \
    if (status_ != HIPBLAS_STATUS_SUCCESS) { \
      fprintf(stderr, "CUBLAS error (%s:%d): %s, %s\n", __FILE__, __LINE__, cublasGetStatusName(status_), cublasGetStatusString(status_));  \
      exit(EXIT_FAILURE); \
    } \
  } while (0)

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;
static hipblasHandle_t handle;

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  // A_gpu = A^T (K X M)
  CHECK_CUBLAS(hipblasSetMatrix(K, M, sizeof(float), _A, K, A_gpu, K));
  // B_gpu = B^T (N X K)
  CHECK_CUBLAS(hipblasSetMatrix(N, K, sizeof(float), _B, N, B_gpu, N));
  // C_gpu = C^T = B^T * A^T (N X M)
  const float alpha = 1.0f, beta = 0.0f;
  CHECK_CUBLAS(hipblasSgemm(handle, HIPBLAS_OP_N, HIPBLAS_OP_N, N, M, K, &alpha, B_gpu, N, A_gpu, K, &beta, C_gpu, N));
  // C = C^T^T = C_gpu^T (M X N)
  CHECK_CUBLAS(hipblasGetMatrix(N, M, sizeof(float), C_gpu, N, _C, N));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc((void **) &A_gpu, sizeof(float) * M * K));
  CHECK_CUDA(hipMalloc((void **) &B_gpu, sizeof(float) * K * N));
  CHECK_CUDA(hipMalloc((void **) &C_gpu, sizeof(float) * M * N));
  CHECK_CUBLAS(hipblasCreate(&handle));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));
  CHECK_CUBLAS(hipblasDestroy(handle));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"

#define BLOCKS 8

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Define kernel function
__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N, int K) {
  int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int y_idx = blockIdx.y * blockDim.y + threadIdx.y;
  if (x_idx >= N || y_idx >= M) return;
  C[y_idx * M + x_idx] = 0.0f;
  float a0, a1, a2, a3, a4, a5, a6, a7;
  float b0, b1, b2, b3, b4, b5, b6, b7;
  int k;

  for (k = 0; k < (K/8) * 8; k+=8) {
    a0 = A[y_idx * K + k];
    a1 = A[y_idx * K + k + 1];
    a2 = A[y_idx * K + k + 2];
    a3 = A[y_idx * K + k + 3];
    a4 = A[y_idx * K + k + 4];
    a5 = A[y_idx * K + k + 5];
    a6 = A[y_idx * K + k + 6];
    a7 = A[y_idx * K + k + 7];

    b0 = B[k * N + x_idx];
    b1 = B[(k + 1) * N + x_idx];
    b2 = B[(k + 2) * N + x_idx];
    b3 = B[(k + 3) * N + x_idx];
    b4 = B[(k + 4) * N + x_idx];
    b5 = B[(k + 5) * N + x_idx];
    b6 = B[(k + 6) * N + x_idx];
    b7 = B[(k + 7) * N + x_idx];

    C[y_idx * M + x_idx] += a0 * b0 + a1 * b1 + a2 * b2 + a3 * b3 + a4 * b4 + a5 * b5 + a6 * b6 + a7 * b7;
  }
  for (; k < K; k++) {
    C[y_idx * M + x_idx] += A[y_idx * K + k] * B[k * N + x_idx];
  }
}

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void naive_cpu_matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int k = 0; k < K; k++) {
      for (int j = 0; j < N; j++) {
        _C[i * N + j] += _A[i * K + k] * _B[k * N + j];
      }
    }
  }
}

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  // // Remove this line after you complete the matmul on GPU
  // naive_cpu_matmul(_A, _B, _C, M, N, K);
  hipStream_t data_stream, calc_stream;
  CHECK_CUDA(hipStreamCreate(&data_stream));
  CHECK_CUDA(hipStreamCreate(&calc_stream));

  // hipEvent_t start, stop;
  // CHECK_CUDA(hipEventCreate(&start));
  // CHECK_CUDA(hipEventCreate(&stop));
  hipEvent_t events[BLOCKS];
  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipEventCreate(&events[i]));
  }
  int Mbegin[BLOCKS], Mend[BLOCKS];
  for (int i = 0; i < BLOCKS; i++) {
    Mbegin[i] = M / BLOCKS * i;
    Mend[i] = M / BLOCKS * (i + 1);
    if (i == BLOCKS - 1) Mend[BLOCKS-1] = M;
  }

  // (TODO) Upload A and B matrix to GPU
  // CHECK_CUDA(hipEventRecord(start, data_stream));
  CHECK_CUDA(hipMemcpyAsync(B_gpu, _B, sizeof(float) * K * N, hipMemcpyHostToDevice, data_stream));
  // CHECK_CUDA(hipMemcpyAsync(A_gpu, _A, sizeof(float) * M * K, hipMemcpyHostToDevice, data_stream));
  
  for (int i=0; i<BLOCKS; i++) {
    CHECK_CUDA(hipMemcpyAsync(A_gpu + Mbegin[i] * K, _A + Mbegin[i] * K, sizeof(float) * (Mend[i] - Mbegin[i]) * K, hipMemcpyHostToDevice, data_stream));
    CHECK_CUDA(hipEventRecord(events[i], data_stream));
  }

  // CHECK_CUDA(hipMemcpyAsync(A_gpu, _A, sizeof(float) * M * K, hipMemcpyHostToDevice, data_stream));
  // CHECK_CUDA(hipEventRecord(stop, data_stream));

  // float data_time;
  // CHECK_CUDA(hipStreamSynchronize(data_stream));
  // CHECK_CUDA(hipEventElapsedTime(&data_time, start, stop));
  // printf("Data transfer time: %f ms\n", data_time);

  // (TODO) Launch kernel on a GPU
  // CHECK_CUDA(hipStreamWaitEvent(calc_stream, stop, 0));
  // CHECK_CUDA(hipEventRecord(start, calc_stream));

  for (int i=0; i<BLOCKS; i++) {
    dim3 blockDim(32, 32);
    dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (Mend[i] - Mbegin[i] + blockDim.y - 1) / blockDim.y);
    CHECK_CUDA(hipStreamWaitEvent(calc_stream, events[i], 0));
    matmul_kernel<<<gridDim, blockDim, 0, calc_stream>>>(&A_gpu[Mbegin[i] * K], B_gpu, &C_gpu[Mbegin[i] * N], Mend[i] - Mbegin[i], N, K);
  }

  // dim3 blockDim(32, 32);
  // dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
  // matmul_kernel<<<gridDim, blockDim, 0, calc_stream>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  // CHECK_CUDA(hipEventRecord(stop, calc_stream));
  // float calc_time;
  // CHECK_CUDA(hipStreamSynchronize(calc_stream));
  // CHECK_CUDA(hipEventElapsedTime(&calc_time, start, stop));
  // printf("Calculation time: %f ms\n", calc_time);

  // (TODO) Download C matrix from GPU
  // CHECK_CUDA(hipStreamWaitEvent(data_stream, stop, 0));
  // CHECK_CUDA(hipEventRecord(start, data_stream));
  CHECK_CUDA(hipStreamSynchronize(calc_stream));
  CHECK_CUDA(hipMemcpyAsync(_C, C_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost, data_stream));
  // CHECK_CUDA(hipEventRecord(stop, data_stream));
  // float download_time;
  // CHECK_CUDA(hipStreamSynchronize(data_stream));
  // CHECK_CUDA(hipEventElapsedTime(&download_time, start, stop));
  // printf("Download time: %f ms\n", download_time);
  // for (int i=1; i<BLOCKS; i++) {
  //   float time;
  //   CHECK_CUDA(hipEventElapsedTime(&time, events[i-1], events[i]));
  //   printf("Block %d time: %f ms\n", i, time);
  // }

  for (int i = 0; i < BLOCKS; i++) {
    CHECK_CUDA(hipEventDestroy(events[i]));
  }
  CHECK_CUDA(hipStreamDestroy(data_stream));
  CHECK_CUDA(hipStreamDestroy(calc_stream));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&A_gpu, sizeof(float) * M * K));
  CHECK_CUDA(hipMalloc(&B_gpu, sizeof(float) * K * N));
  CHECK_CUDA(hipMalloc(&C_gpu, sizeof(float) * M * N));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

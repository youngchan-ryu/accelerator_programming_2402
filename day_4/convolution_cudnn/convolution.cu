#include <cstdio>
#include <stdlib.h>
#include <hipDNN.h>

#include "convolution.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

#define CHECK_CUDNN(call)                                                 \
  do {                                                                   \
    hipdnnStatus_t status_ = call;                                          \
    if (status_ != HIPDNN_STATUS_SUCCESS) {                                        \
      fprintf(stderr, "CUDNN error (%s:%d): %s\n", __FILE__, __LINE__, \
              hipdnnGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

static hipdnnHandle_t handle;
static hipdnnTensorDescriptor_t input_desc;
static hipdnnFilterDescriptor_t filter_desc;
static hipdnnConvolutionDescriptor_t conv_desc;
static hipdnnTensorDescriptor_t output_desc;
static int ON, OC, OH, OW;
static float *I_gpu, *F_gpu, *O_gpu, *workspace;
static hipdnnConvolutionFwdAlgoPerf_t best_algo;

static const char *algo_to_string(hipdnnConvolutionFwdAlgo_t algo);

void convolution(float *_I, float *_F, float *_O, int N, int C, int H, int W,
                 int K, int R, int S, int pad_h, int pad_w, int stride_h,
                 int stride_w, int dilation_h, int dilation_w) {
  CHECK_CUDA(hipMemcpy(I_gpu, _I, sizeof(float) * N * C * H * W, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(F_gpu, _F, sizeof(float) * K * C * R * S, hipMemcpyHostToDevice));

  const float alpha = 1.0f, beta = 0.0f;
  CHECK_CUDNN(hipdnnConvolutionForward(
      handle, &alpha, input_desc, I_gpu, filter_desc, F_gpu, conv_desc,
      best_algo.algo, workspace, best_algo.memory, &beta, output_desc, O_gpu));

  CHECK_CUDA(hipMemcpy(_O, O_gpu, sizeof(float) * ON * OC * OH * OW, hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void convolution_initialize(int N, int C, int H, int W, int K, int R, int S,
                            int pad_h, int pad_w, int stride_h, int stride_w,
                            int dilation_h, int dilation_w) {
  CHECK_CUDNN(hipdnnCreate(&handle));

  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&input_desc));
  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(input_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, N, C, H, W));

  CHECK_CUDNN(hipdnnCreateFilterDescriptor(&filter_desc));
  CHECK_CUDNN(hipdnnSetFilter4dDescriptor(filter_desc, HIPDNN_DATA_FLOAT, HIPDNN_TENSOR_NCHW, K, C, R, S));

  CHECK_CUDNN(hipdnnCreateConvolutionDescriptor(&conv_desc));
  CHECK_CUDNN(hipdnnSetConvolution2dDescriptor(conv_desc, pad_h, pad_w, stride_h, stride_w, dilation_h, dilation_w, HIPDNN_CROSS_CORRELATION, HIPDNN_DATA_FLOAT));

  CHECK_CUDNN(hipdnnGetConvolution2dForwardOutputDim(conv_desc, input_desc, filter_desc, &ON, &OC, &OH, &OW));

  CHECK_CUDNN(hipdnnCreateTensorDescriptor(&output_desc));
  CHECK_CUDNN(hipdnnSetTensor4dDescriptor(output_desc, HIPDNN_TENSOR_NCHW, HIPDNN_DATA_FLOAT, ON, OC, OH, OW));

  int max_algo_count;
  CHECK_CUDNN(cudnnGetConvolutionForwardAlgorithmMaxCount(handle, &max_algo_count));

  int returned_algo_count;
  hipdnnConvolutionFwdAlgoPerf_t algo_perfs[max_algo_count];
  CHECK_CUDNN(hipdnnFindConvolutionForwardAlgorithm(handle, input_desc, filter_desc, conv_desc, output_desc, max_algo_count, &returned_algo_count, algo_perfs));

  for (int i=0; i<returned_algo_count; ++i) {
    printf("Algorithm %d: name %s, time %f sec, memory %lu byte, status %s\n", i, algo_to_string(algo_perfs[i].algo), algo_perfs[i].time, algo_perfs[i].memory, hipdnnGetErrorString(algo_perfs[i].status));
  }

  best_algo = algo_perfs[0];

  CHECK_CUDA(hipMalloc(&I_gpu, sizeof(float) * N * C * H * W));
  CHECK_CUDA(hipMalloc(&F_gpu, sizeof(float) * K * C * R * S));
  CHECK_CUDA(hipMalloc(&O_gpu, sizeof(float) * ON * OC * OH * OW));
  CHECK_CUDA(hipMalloc(&workspace, algo_perfs[0].memory));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void convolution_cleanup(float *_I, float *_F, float *_O, int N, int C, int H,
                         int W, int K, int R, int S, int pad_h, int pad_w,
                         int stride_h, int stride_w, int dilation_h,
                         int dilation_w) {

  CHECK_CUDA(hipFree(I_gpu));
  CHECK_CUDA(hipFree(F_gpu));
  CHECK_CUDA(hipFree(O_gpu));
  CHECK_CUDA(hipFree(workspace));

  CHECK_CUDNN(hipdnnDestroyTensorDescriptor(input_desc));
  CHECK_CUDNN(hipdnnDestroyFilterDescriptor(filter_desc));
  CHECK_CUDNN(hipdnnDestroyConvolutionDescriptor(conv_desc));
  CHECK_CUDNN(hipdnnDestroyTensorDescriptor(output_desc));
  CHECK_CUDNN(hipdnnDestroy(handle));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

const char *algo_to_string(hipdnnConvolutionFwdAlgo_t algo) {
  switch (algo) {
    case HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_GEMM";
    case HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_IMPLICIT_PRECOMP_GEMM";
    case HIPDNN_CONVOLUTION_FWD_ALGO_GEMM:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_GEMM";
    case HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_DIRECT";
    case HIPDNN_CONVOLUTION_FWD_ALGO_FFT:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_FFT";
    case HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_FFT_TILING";
    case HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD";
    case HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_WINOGRAD_NONFUSED";
    case HIPDNN_CONVOLUTION_FWD_ALGO_COUNT:
      return "HIPDNN_CONVOLUTION_FWD_ALGO_COUNT";
    default: return "<unknown algorithm>";
  }
}
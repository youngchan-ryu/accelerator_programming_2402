#include "hip/hip_runtime.h"
#include <cstdio>

#include "image_rotation.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Define kernel function
__global__ void rotate_image_kernel(float *input_images, float *output_images, int W, int H, float sin_theta, float cos_theta, int i) {
  int dest_x = blockIdx.x * blockDim.x + threadIdx.x;
  int dest_y = blockIdx.y * blockDim.y + threadIdx.y;
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;
  if(dest_x >= W || dest_y >= H) return;
  float xOff = dest_x - x0;
  float yOff = dest_y - y0;
  int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
  int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
  if((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
    output_images[i * H * W + dest_y * W + dest_x] = input_images[i * H * W + src_y * W + src_x];
  } else {
    output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
  }
}

__global__ void rotate_image_kernel_3(float *input_images, float *output_images, int W, int H, float sin_theta, float cos_theta, int N) {
  int n = blockIdx.z * blockDim.z + threadIdx.z;
  int dest_x = blockIdx.x * blockDim.x + threadIdx.x;
  int dest_y = blockIdx.y * blockDim.y + threadIdx.y;
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;
  if(dest_x >= W || dest_y >= H) return;
  float xOff = dest_x - x0;
  float yOff = dest_y - y0;
  int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
  int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
  if((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
    output_images[n * H * W + dest_y * W + dest_x] = input_images[n * H * W + src_y * W + src_x];
  } else {
    output_images[n * H * W + dest_y * W + dest_x] = 0.0f;
  }
}


// Device(GPU) pointers
static float *input_images_gpu, *output_images_gpu;

void rotate_image_naive(float *input_images, float *output_images, int W, int H,
                        float sin_theta, float cos_theta, int num_src_images) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  for (int i = 0; i < num_src_images; i++) {
    for (int dest_x = 0; dest_x < W; dest_x++) {
      for (int dest_y = 0; dest_y < H; dest_y++) {
        float xOff = dest_x - x0;
        float yOff = dest_y - y0;
        int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
        int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
        if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
          output_images[i * H * W + dest_y * W + dest_x] =
              input_images[i * H * W + src_y * W + src_x];
        } else {
          output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
        }
      }
    }
  }
}

void rotate_image(float *input_images, float *output_images, int W, int H,
                  float sin_theta, float cos_theta, int num_src_images) {
  // // Remove this line after you complete the image rotation on GPU
  // rotate_image_naive(input_images, output_images, W, H, sin_theta, cos_theta,
  //                    num_src_images);

  // (TODO) Upload input images to GPU
  CHECK_CUDA(hipMemcpy(input_images_gpu, input_images, sizeof(float) * W * H * num_src_images, hipMemcpyHostToDevice));

  // (TODO) Launch kernel on GPU
  // dim3 block(16, 16);
  // dim3 grid((W + block.x - 1) / block.x, (H + block.y - 1) / block.y);
  // for (int i = 0; i < num_src_images; i++) {
  //   rotate_image_kernel<<<grid, block>>>(input_images_gpu, output_images_gpu, W, H, sin_theta, cos_theta, i);
  // }
  dim3 blockDim3(32,32,1);
  dim3 gridDim3((W + blockDim3.x - 1) / blockDim3.x, (H + blockDim3.y - 1) / blockDim3.y, num_src_images);
  rotate_image_kernel_3<<<gridDim3, blockDim3>>>(input_images_gpu, output_images_gpu, W, H, sin_theta, cos_theta, num_src_images);

  // (TODO) Download output images from GPU
  CHECK_CUDA(hipMemcpy(output_images, output_images_gpu, sizeof(float) * W * H * num_src_images, hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_init(int image_width, int image_height, int num_src_images) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&input_images_gpu, sizeof(float) * image_width * image_height * num_src_images));
  CHECK_CUDA(hipMalloc(&output_images_gpu, sizeof(float) * image_width * image_height * num_src_images));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_cleanup() {
  // (TODO) Free device memory
  CHECK_CUDA(hipFree(input_images_gpu));
  CHECK_CUDA(hipFree(output_images_gpu));

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}


#include <hip/hip_runtime.h>
#include <chrono>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

int main() {
  int bytes = sizeof(int) * (1 << 20);

  int *d_a;
  CHECK_CUDA(hipMalloc(&d_a, bytes));

  /* 1. Pageable memory test */
  {
    int *a_pageable;
    // TODO: Allocate pageable memory using malloc
    auto start = std::chrono::system_clock::now();
    // TODO: Run H2D memcpy on pageable memory
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = end - start;
    printf("Pageable memory bandwidth: %lf GB/s\n",
           (bytes / diff.count() / 1000. / 1e9));
  }

  /* 2. Pinned memory test */
  {
    int *a_pinned;
    // TODO: Allocate pinned memory using cudaMallocHost
    auto start = std::chrono::system_clock::now();
    // TODO: Run H2D memcpy on pinned memory
    auto end = std::chrono::system_clock::now();
    std::chrono::duration<double> diff = end - start;
    printf("Pinned memory bandwidth: %lf GB/s\n",
           (bytes / diff.count() / 1000. / 1e9));
  }
  return 0;
}

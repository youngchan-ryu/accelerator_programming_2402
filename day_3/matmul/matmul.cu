#include "hip/hip_runtime.h"
#include <cstdio>

#include "matmul.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Define kernel function
__global__ void matmul_kernel(float *A, float *B, float *C, int M, int N, int K) {
  int x_idx = blockIdx.x * blockDim.x + threadIdx.x;
  int y_idx = blockIdx.y * blockDim.y + threadIdx.y;
  if (x_idx >= N || y_idx >= M) return;
  C[y_idx * N + x_idx] = 0.0f;
  for (int k = 0; k < K; k++) {
    C[y_idx * N + x_idx] += A[y_idx * K + k] * B[k * N + x_idx];
  }
}

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void naive_cpu_matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int k = 0; k < K; k++) {
      for (int j = 0; j < N; j++) {
        _C[i * N + j] += _A[i * K + k] * _B[k * N + j];
      }
    }
  }
}

void matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  // // Remove this line after you complete the matmul on GPU
  // naive_cpu_matmul(_A, _B, _C, M, N, K);

  // (TODO) Upload A and B matrix to GPU
  CHECK_CUDA(hipMemcpy(A_gpu, _A, sizeof(float) * M * K, hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(B_gpu, _B, sizeof(float) * K * N, hipMemcpyHostToDevice));

  // (TODO) Launch kernel on a GPU
  dim3 blockDim(32, 32);
  dim3 gridDim((N + blockDim.x - 1) / blockDim.x, (M + blockDim.y - 1) / blockDim.y);
  matmul_kernel<<<gridDim, blockDim>>>(A_gpu, B_gpu, C_gpu, M, N, K);

  // (TODO) Download C matrix from GPU
  CHECK_CUDA(hipMemcpy(_C, C_gpu, sizeof(float) * M * N, hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&A_gpu, sizeof(float) * M * K));
  CHECK_CUDA(hipMalloc(&B_gpu, sizeof(float) * K * N));
  CHECK_CUDA(hipMalloc(&C_gpu, sizeof(float) * M * N));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(float *_A, float *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

#include <cstdio>
#include <mma.h>

#include "matmul.h"

using namespace nvcuda;

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void naive_cpu_matmul(half *_A, half *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int k = 0; k < K; k++) {
      for (int j = 0; j < N; j++) {
        _C[i * N + j] += (float)_A[i * K + k] * (float)_B[k * N + j];
      }
    }
  }
}

void matmul(half *_A, half *_B, float *_C, int M, int N, int K) {
  // Remove this line after you complete the matmul on GPU
  naive_cpu_matmul(_A, _B, _C, M, N, K);

  // (TODO) Upload A and B matrix to GPU

  // (TODO) Launch kernel on a GPU

  // (TODO) Download C matrix from GPU

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_init(int M, int N, int K) {
  // (TODO) Allocate device memory

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void matmul_cleanup(half *_A, half *_B, float *_C, int M, int N, int K) {
  // (TODO) Do any post-matmul cleanup work here.

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

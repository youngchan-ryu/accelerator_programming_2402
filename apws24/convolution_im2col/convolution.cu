#include <cstdlib>

#include "convolution.cuh"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

void naive_cpu_im2col(float *_I, float *workspace, int N, int C, int H, int W,
                      int R, int S, int pad_h, int pad_w, int stride_h,
                      int stride_w, int dilation_h, int dilation_w) {
  float *I = _I;

  // Naive CPU im2col
  const int ON = N;
  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;

  for (int on = 0; on < ON; ++on) {
    for (int oh = 0; oh < OH; ++oh) {
      for (int ow = 0; ow < OW; ++ow) {
        for (int c = 0; c < C; ++c) {
          for (int r = 0; r < R; ++r) {
            for (int s = 0; s < S; ++s) {
              const int n = on;
              const int h = oh * stride_h - pad_h + r * dilation_h;
              const int w = ow * stride_w - pad_w + s * dilation_w;

              if (h < 0 || h >= H || w < 0 || w >= W) continue;

              workspace[(((size_t)c * R * S) + (r * S) + s) * (ON * OH * OW) +
                        (on * OH * OW + oh * OW + ow)] =
                  I[(size_t)n * C * H * W + c * H * W + h * W + w];
            }
          }
        }
      }
    }
  }
}

void naive_cpu_matmul(float *_A, float *_B, float *_C, int M, int N, int K) {
  for (int i = 0; i < M; i++) {
    for (int k = 0; k < K; k++) {
      for (int j = 0; j < N; j++) {
        _C[(size_t)i * N + j] += _A[(size_t)i * K + k] * _B[(size_t)k * N + j];
      }
    }
  }
}

void reshape(float *_src, float *_dst, int N, int K, int OH, int OW) {
  size_t chunk = OH * OW;

  for (int on = 0; on < N; ++on) {
    for (int k = 0; k < K; ++k) {
      memcpy((void *) (_dst + (((size_t)on * K + k) * chunk)),
             (void *) (_src + (((size_t)k * N + on) * chunk)), chunk * sizeof(float));
    }
  }
}

void naive_cpu_convolution_im2col(float *_I, float *_F, float *_O, float *_BUF1,
                                  float *_BUF2, int N, int C, int H, int W,
                                  int K, int R, int S, int pad_h, int pad_w,
                                  int stride_h, int stride_w, int dilation_h,
                                  int dilation_w) {
  float *I = _I, *F = _F, *O = _O, *BUF1 = _BUF1, *BUF2 = _BUF2;

  const int OH = 1 + (H + 2 * pad_h - (((R - 1) * dilation_h) + 1)) / stride_h;
  const int OW = 1 + (W + 2 * pad_w - (((S - 1) * dilation_w) + 1)) / stride_w;

  naive_cpu_im2col(I, BUF1, N, C, H, W, R, S, pad_h, pad_w, stride_h, stride_w,
                   dilation_h, dilation_w);

  naive_cpu_matmul(F, BUF1, BUF2, K, N * OH * OW, C * R * S);

  reshape(BUF2, O, N, K, OH, OW);
}

void convolution(float *_I, float *_F, float *_O, float *_BUF1, float *_BUF2,
                 int N, int C, int H, int W, int K, int R, int S, int pad_h,
                 int pad_w, int stride_h, int stride_w, int dilation_h,
                 int dilation_w) {
  // Remove this line after you complete the convolution on GPU
  naive_cpu_convolution_im2col(_I, _F, _O, _BUF1, _BUF2, N, C, H, W, K, R, S,
                               pad_h, pad_w, stride_h, stride_w, dilation_h,
                               dilation_w);
}

void convolution_initialize(int N, int C, int H, int W, int K, int R, int S,
                            int pad_h, int pad_w, int stride_h, int stride_w,
                            int dilation_h, int dilation_w) {}

void convolution_cleanup(float *_I, float *_F, float *_O, int N, int C, int H,
                         int W, int K, int R, int S, int pad_h, int pad_w,
                         int stride_h, int stride_w, int dilation_h,
                         int dilation_w) {}
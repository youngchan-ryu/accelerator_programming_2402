#include "hip/hip_runtime.h"
#include <cstdio>

#include "image_rotation.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

// Device(GPU) pointers
static float *input_images_gpu, *output_images_gpu;

void rotate_image_naive(float *input_images, float *output_images, int W, int H,
                        float sin_theta, float cos_theta, int num_src_images) {
  float x0 = W / 2.0f;
  float y0 = H / 2.0f;

  // Rotate images
  for (int i = 0; i < num_src_images; i++) {
    for (int dest_x = 0; dest_x < W; dest_x++) {
      for (int dest_y = 0; dest_y < H; dest_y++) {
        float xOff = dest_x - x0;
        float yOff = dest_y - y0;
        int src_x = (int) (xOff * cos_theta + yOff * sin_theta + x0);
        int src_y = (int) (yOff * cos_theta - xOff * sin_theta + y0);
        if ((src_x >= 0) && (src_x < W) && (src_y >= 0) && (src_y < H)) {
          output_images[i * H * W + dest_y * W + dest_x] =
              input_images[i * H * W + src_y * W + src_x];
        } else {
          output_images[i * H * W + dest_y * W + dest_x] = 0.0f;
        }
      }
    }
  }
}

void rotate_image(float *input_images, float *output_images, int W, int H,
                  float sin_theta, float cos_theta, int num_src_images) {
  // Remove this line after you complete the image rotation on GPU
  rotate_image_naive(input_images, output_images, W, H, sin_theta, cos_theta,
                     num_src_images);

  // (TODO) Upload input images to GPU

  // (TODO) Launch kernel on GPU

  // (TODO) Download output images from GPU

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_init(int image_width, int image_height, int num_src_images) {
  // (TODO) Allocate device memory

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void rotate_image_cleanup() {
  // (TODO) Free device memory

  // DO NOT REMOVE; NEED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}


#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

int main() {
  int count;
  CHECK_CUDA(hipGetDeviceCount(&count));

  printf("Number of devices: %d\n", count);
  hipDeviceProp_t props[4];
  for (int i = 0; i < count; ++i) {
    printf("\tdevice %d:\n", i);
    // TODO: get and print device properties
  }

  return 0;
}


#include <hip/hip_runtime.h>
#include <cstdio>

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

int main() {
  int count;
  CHECK_CUDA(hipGetDeviceCount(&count));

  printf("Number of devices: %d\n", count);
  hipDeviceProp_t props[4];
  for (int i = 0; i < count; ++i) {
    printf("\tdevice %d:\n", i);
    // TODO: get and print device properties
    CHECK_CUDA(hipGetDeviceProperties(&props[i], i));
    printf("\t\tname: %s\n", props[i].name);
    printf("\t\tmultiProcessorCount: %d\n", props[i].multiProcessorCount);
    printf("\t\tmaxThreadsPerBlock: %d\n", props[i].maxThreadsPerBlock);
    printf("\t\ttotalGlobalMem: %zu\n", props[i].totalGlobalMem);
    printf("\t\tsharedMemPerBlock: %zu\n", props[i].sharedMemPerBlock);
  }

  return 0;
}

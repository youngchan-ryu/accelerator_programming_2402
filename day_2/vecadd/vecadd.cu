#include "hip/hip_runtime.h"
#include <cstdio>

#include "vecadd.h"

#define CHECK_CUDA(call)                                                 \
  do {                                                                   \
    hipError_t status_ = call;                                          \
    if (status_ != hipSuccess) {                                        \
      fprintf(stderr, "CUDA error (%s:%d): %s:%s\n", __FILE__, __LINE__, \
              hipGetErrorName(status_), hipGetErrorString(status_));   \
      exit(EXIT_FAILURE);                                                \
    }                                                                    \
  } while (0)

__global__ void vecadd_kernel(const int N, const float *a, const float *b, float *c) {
  int tidx = blockIdx.x * blockDim.x + threadIdx.x;
  c[tidx] = a[tidx] + b[tidx];
}

// Device(GPU) pointers
static float *A_gpu, *B_gpu, *C_gpu;

void vecadd(float *_A, float *_B, float *_C, int N) {
  // (TODO) Upload A and B vector to GPU
  CHECK_CUDA(hipMemcpy(A_gpu, _A, N * sizeof(float), hipMemcpyHostToDevice));
  CHECK_CUDA(hipMemcpy(B_gpu, _B, N * sizeof(float), hipMemcpyHostToDevice));

  // Launch kernel on a GPU
  dim3 gridDim(N / 512);
  dim3 blockDim(512);
  vecadd_kernel<<<gridDim, blockDim>>>(N, A_gpu, B_gpu, C_gpu);

  // (TODO) Download C vector from GPU
  CHECK_CUDA(hipMemcpy(_C, C_gpu, N * sizeof(float), hipMemcpyDeviceToHost));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void vecadd_init(int N) {
  // (TODO) Allocate device memory
  CHECK_CUDA(hipMalloc(&A_gpu, N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&B_gpu, N * sizeof(float)));
  CHECK_CUDA(hipMalloc(&C_gpu, N * sizeof(float)));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}

void vecadd_cleanup(float *_A, float *_B, float *_C, int N) {
  // (TODO) Do any post-vecadd cleanup work here.
  CHECK_CUDA(hipFree(A_gpu));
  CHECK_CUDA(hipFree(B_gpu));
  CHECK_CUDA(hipFree(C_gpu));

  // DO NOT REMOVE; NEEDED FOR TIME MEASURE
  CHECK_CUDA(hipDeviceSynchronize());
}


#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_world() {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  printf("Device(GPU) Thread %d: Hello, World!\n", tidx);
}

int main()
{
  hello_world<<<2, 4>>>();
  hipDeviceSynchronize();
  return 0;
}

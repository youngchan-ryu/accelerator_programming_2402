
#include <hip/hip_runtime.h>
#include <cstdio>

__global__ void hello_world() {
  int tidx = threadIdx.x + blockIdx.x * blockDim.x;
  printf("Device(GPU) Thread %d: Hello, World!\n", tidx);
}

int main()
{
  hello_world<<<4, 8>>>();
  hipDeviceSynchronize();
  return 0;
}
